#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "cuda-error.h"

#define imin(a, b) (a < b ? a : b)

// Number of float numbers (36 millions).
const int N = 33 * 1024 * 1024;

// Number of threads for each block.
const int THREADS = 256;

// By default there are 32 blocks, but if the source array is shorter there may
// be fewer than 32 blocks.
const int BLOCKS = imin(32, (N + THREADS - 1) / THREADS );

__global__ void dot(float *a, float *b, float *c) {
    __shared__ float cache[THREADS];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float tmp = 0;
    while (tid < N) {
        tmp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // Set cache value.
    cache[cacheIndex] = tmp;

    // Sync threads in the block.
    __syncthreads();

    // For reductions, threadsPerBlock must be a power of 2 because of the
    // following code
    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];

        __syncthreads();
        i /= 2;
    }

    // Only one thread of the block will write to c.
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

float malloc_test(size_t size, float *result) {
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;
    float elapsed;

    // Create event to measure elapsed time.
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    // Allocate memory host side.
    a = (float *) malloc(sizeof(float) * size);
    b = (float *) malloc(sizeof(float) * size);
    partial_c = (float *) malloc(sizeof(float) * BLOCKS);

    // Allocate memory device side.
    HANDLE_ERROR(hipMalloc((void **)&dev_a, sizeof(float) * size));
    HANDLE_ERROR(hipMalloc((void **)&dev_b, sizeof(float) * size));
    HANDLE_ERROR(hipMalloc((void **)&dev_partial_c,
                sizeof(float) * BLOCKS));

    // dev_partial_c and partial_c stores the partial results, one for each
    // block, this is why it is large sizeof(float) * BLOCKS.

    // Fill the input arrays.
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    // Start timer.
    HANDLE_ERROR(hipEventRecord(start));

    // Copy input array to the device.
    HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(float) * size,
                hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(float) * size,
                hipMemcpyHostToDevice));

    // Run kernel.
    dot<<<BLOCKS, THREADS>>>(dev_a, dev_b, dev_partial_c);

    // Copy output array to host.
    HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c,
                sizeof(float) * BLOCKS, hipMemcpyDeviceToHost));

    // Stop timer.
    HANDLE_ERROR(hipEventRecord(stop));
    HANDLE_ERROR(hipEventSynchronize(stop));
    HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));

    // Make last reduction on host.
    c = 0;
    for (int i = 0; i < BLOCKS; i++)
        c += partial_c[i];
    *result = c;

    // Free memory both host and device side.
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_partial_c));
    free(a);
    free(b);
    free(partial_c);

    return elapsed;
}

float mapped_memory_test(size_t size, float *result) {
    // Create event to measure elapsed time.
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    // Allocate mapped page-locked memory on the host.
    float *a, *b, *partial_c;
    HANDLE_ERROR(hipHostAlloc((void **)&a, size * sizeof(float),
                hipHostMallocMapped | hipHostMallocWriteCombined));
    HANDLE_ERROR(hipHostAlloc((void **)&b, size * sizeof(float),
                hipHostMallocMapped | hipHostMallocWriteCombined));
    // partial_c stores the partial results, one for each block, this is why it
    // is large sizeof(float) * BLOCKS.
    HANDLE_ERROR(hipHostAlloc((void **)&partial_c, BLOCKS * sizeof(float),
                hipHostMallocMapped));

    // Fill the input arrays.
    for (int i = 0; i < size; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    // Get pointers in the device for mapped host memory.
    float *dev_a, *dev_b, *dev_partial_c;
    HANDLE_ERROR(hipHostGetDevicePointer((void **)&dev_a, a, 0));
    HANDLE_ERROR(hipHostGetDevicePointer((void **)&dev_b, b, 0));
    HANDLE_ERROR(hipHostGetDevicePointer((void **)&dev_partial_c, partial_c, 0));

    // Start timer.
    HANDLE_ERROR(hipEventRecord(start));

    // Run kernel.
    dot<<<BLOCKS, THREADS>>>(dev_a, dev_b, dev_partial_c);

    // Stop timer.
    HANDLE_ERROR(hipEventRecord(stop));
    HANDLE_ERROR(hipEventSynchronize(stop));
    float elapsed;
    HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));

    // Make last reduction on host. Because we synchronized with the device, we
    // are sure to read correct value in partial_c.
    float c = 0;
    for (int i = 0; i < BLOCKS; i++)
        c += partial_c[i];
    *result = c;

    // Free memory both host and device side.
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
    HANDLE_ERROR(hipHostFree(a));
    HANDLE_ERROR(hipHostFree(b));
    HANDLE_ERROR(hipHostFree(partial_c));

    return elapsed;
}

int main() {
    // Get current device.
    int device;
    HANDLE_ERROR(hipGetDevice(&device));

    // Device must support mapped page-locked host memory.
    hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, device));
    if (prop.canMapHostMemory != 1) {
        puts("Device cannot map host memory into the CUDA address space.");
        return EXIT_FAILURE;
    }

    // Enable mapped memory.
    HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));

    float result;
    float elapsed = malloc_test(N, &result);
    printf("Result with hipMalloc: %.6g\n", result);
    printf("Time using hipMalloc: %3.1f ms\n", elapsed);

    elapsed = mapped_memory_test(N, &result);
    printf("Result with cudaHostMalloc: %.6g\n", result);
    printf("Time using cudaHostMalloc: %3.1f ms\n", elapsed);

    return EXIT_SUCCESS;
}
