#include <stdlib.h>
#include <stdio.h>

#include "cuda-error.h"

// Number of hipMemcpy operations make in each test.
const unsigned int COPIES = 100;

// Size of data.
const size_t SIZE = 4 * 1024 * 1024; // 10 MB.

// Returns the time (in ms) to make several dummy hipMemcpy copies with data of
// the given size allocated with malloc and in the given copy direction.
float cuda_malloc_test(int size, bool copyToDevice) {
    // Create events uset to track the time of the copies.
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    // Allocate host data.
    int *data = (int *)malloc(size * sizeof(data));
    HANDLE_NULL(data);

    // Allocate device data.
    int *dev_data;
    HANDLE_ERROR(hipMalloc((void **)&dev_data, size * sizeof(data)));

    // Track the events and do the copies.
    HANDLE_ERROR(hipEventRecord(start));
    for (unsigned int i = 0; i < COPIES; i++) {
        if (copyToDevice)
            HANDLE_ERROR(hipMemcpy(dev_data, data, size * sizeof(data),
                        hipMemcpyHostToDevice));
        else
            HANDLE_ERROR(hipMemcpy(data, dev_data, size * sizeof(data),
                        hipMemcpyDeviceToHost));
    }
    HANDLE_ERROR(hipEventRecord(stop));
    HANDLE_ERROR(hipEventSynchronize(stop));

    // Get elapsed time.
    float elapsed;
    HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));

    // Free all allocated data.
    HANDLE_ERROR(hipFree(dev_data));
    free(data);
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    return elapsed;
}

// Returns the time (in ms) to make several dummy hipMemcpy copies with data of
// the given size allocated with hipHostAlloc and in the given copy direction.
//
// The hipHostAlloc flags can be specified with alloc_flags.
float cuda_host_alloc_test(int size, bool copyToDevice, unsigned int alloc_flags) {
    // Create events uset to track the time of the copies.
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    // Allocate host data.
    int *data;
    HANDLE_ERROR(hipHostAlloc((void **)&data, size * sizeof(data),
                alloc_flags));

    // Allocate device data.
    int *dev_data;
    HANDLE_ERROR(hipMalloc((void **)&dev_data, size * sizeof(data)));

    // Track the events and do the copies.
    HANDLE_ERROR(hipEventRecord(start));
    for (unsigned int i = 0; i < COPIES; i++) {
        if (copyToDevice)
            HANDLE_ERROR(hipMemcpy(dev_data, data, size * sizeof(data),
                        hipMemcpyHostToDevice));
        else
            HANDLE_ERROR(hipMemcpy(data, dev_data, size * sizeof(data),
                        hipMemcpyDeviceToHost));
    }
    HANDLE_ERROR(hipEventRecord(stop));
    HANDLE_ERROR(hipEventSynchronize(stop));

    // Get elapsed time.
    float elapsed;
    HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));

    // Free all allocated data.
    HANDLE_ERROR(hipFree(dev_data));
    HANDLE_ERROR(hipHostFree(data));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    return elapsed;
}

int main() {
    float elapsed;
    float mb = (float)100 * SIZE * sizeof(int) / 1024 / 1024;

    // Benchmark with malloc and host->device copy.
    elapsed = cuda_malloc_test(SIZE, true);
    printf("Time using malloc (host->device): %4.1f ms (%4.1f MB/s)\n", elapsed,
            mb / (elapsed / 1000));

    // Benchmark with malloc and device->host copy.
    elapsed = cuda_malloc_test(SIZE, false);
    printf("Time using malloc (device->host): %4.1f ms (%4.1f MB/s)\n", elapsed,
            mb / (elapsed / 1000));

    // Benchmark with hipHostAlloc with default flag and host->device copy.
    elapsed = cuda_host_alloc_test(SIZE, true, hipHostMallocDefault);
    printf("Time using hipHostAlloc with default flags (host->device): %4.1f ms (%4.1f MB/s)\n",
            elapsed, mb / (elapsed / 1000));

    // Benchmark with hipHostAlloc with default flag and device->host copy.
    elapsed = cuda_host_alloc_test(SIZE, false, hipHostMallocDefault);
    printf("Time using hipHostAlloc with default flags (device->host): %4.1f ms (%4.1f MB/s)\n",
            elapsed, mb / (elapsed / 1000));

    // Benchmark with hipHostAlloc with WriteCombined flag and host->device copy.
    elapsed = cuda_host_alloc_test(SIZE, true, hipHostMallocDefault | hipHostMallocWriteCombined);
    printf("Time using hipHostAlloc with WriteCombined flag (host->device): %4.1f ms (%4.1f MB/s)\n",
            elapsed, mb / (elapsed / 1000));

    return EXIT_SUCCESS;
}
