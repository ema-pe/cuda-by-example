// This is a test file to larn CUDA Array and Texture Object.

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// HandleError, HANDLE_ERROR, HandleNull, HANDLE_NULL are utilies to handle
// errors on CUDA calls and malloc calls.

static void HandleError(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR(err) (HandleError( err, __FILE__, __LINE__ ))

static void HandleNull(void *ptr, const char *file, int line) {
        if (ptr == NULL) {
                printf("Host memory failed in %s at line %d\n", file, line);
                exit(EXIT_FAILURE);
        }
}

#define HANDLE_NULL(a) (HandleNull(a, __FILE__, __LINE__))

// Square dimension of the matrix.
const size_t DIM = 32;

__global__ void set_value_kernel(float *output, hipTextureObject_t input) {
    // Map from threadIdx/BlockIdx to cell position.
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    float value = tex2D<float>(input, x, y);

    output[offset] = value;
}

void init_input_array(hipArray_t dev_input) {
    // Input is a 2D array but is linear on host memory.
    float *input = (float *)malloc(DIM * DIM * sizeof(float));
    HANDLE_NULL(input);

    // Set host array values.
    for (int i = 0; i < DIM * DIM; i++)
        input[i] = i;

    // Copy host array to device array.
    HANDLE_ERROR(hipMemcpy2DToArray(dev_input,
                0, 0, // Offsets (not used).
                input,
                DIM * sizeof(float), // Width in bytes plus padding (no padding).
                DIM * sizeof(float), // Width in bytes.
                DIM, // Number of rows (not bytes).
                hipMemcpyHostToDevice));

    free(input);
}

int main() {
    // Allocate a CUDA 2D array on the device. This array is used to create a
    // texture object that is used as input in the kernel.
    hipChannelFormatDesc fmt_desc;
    memset(&fmt_desc, 0, sizeof(hipChannelFormatDesc));
    fmt_desc.f = hipChannelFormatKindFloat;
    fmt_desc.x = 32;
    hipArray_t dev_input;
    HANDLE_ERROR(hipMallocArray(&dev_input, &fmt_desc, DIM, DIM, hipArraySurfaceLoadStore));

    init_input_array(dev_input); // Initialise the array with default values.

    // Create the resource description of the texture.
    hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(hipResourceDesc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = dev_input;

    // Create the texture description (with default options).
    hipTextureDesc tex_desc;
    memset(&tex_desc, 0, sizeof(hipTextureDesc));

    // Finally create the texture object.
    hipTextureObject_t input_tex;
    HANDLE_ERROR(hipCreateTextureObject(&input_tex, &res_desc, &tex_desc, NULL));

    // Allocate a linear output buffer in the device. It is used as 2D array.
    float *dev_output;
    HANDLE_ERROR(hipMalloc((void **)&dev_output, DIM * DIM * sizeof(float)));

    // Launch the kernel with 4x4 blocks of 8x8 threads for each block. The
    // total is 1024, the dimension of the input (32x32).
    dim3 blocks(DIM / 8, DIM / 8);
    dim3 threads(8, 8);
    set_value_kernel<<<blocks, threads>>>(dev_output, input_tex);

    // Allocate a linear output buffer in the host. It is used as 2D array.
    float *output = (float *)calloc(DIM * DIM, sizeof(float));
    HANDLE_NULL(output);

    // Copy output buffer from device to host.
    HANDLE_ERROR(hipMemcpy(output, dev_output, DIM * DIM * sizeof(float),
                hipMemcpyDeviceToHost));

    // Print output.
    for (int row = 0; row < DIM; row++) {
        for (int column = 0; column < DIM; column++) {
            printf("%5.0f ", output[column + row * DIM]);
        }
        puts("");
    }

    // Free allocated memory.
    HANDLE_ERROR(hipFree(dev_output));
    HANDLE_ERROR(hipDestroyTextureObject(input_tex));
    HANDLE_ERROR(hipFreeArray(dev_input));
    free(output);

    return EXIT_SUCCESS;
}
