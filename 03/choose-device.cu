#include <stdio.h>
#include <string.h>

#include "../common/cuda-error.h"

int main(void) {
    hipDeviceProp_t prop;
    int device;

    HANDLE_ERROR(hipGetDevice(&device));
    printf("ID of current CUDA device: %d\n", device);

    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 7;
    prop.minor = 0;

    HANDLE_ERROR(hipChooseDevice(&device, &prop));

    printf("ID of the closest CUDA device: %d\n", device);
    HANDLE_ERROR(hipSetDevice(device));

    return 0;
}
