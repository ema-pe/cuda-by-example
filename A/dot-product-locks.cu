#include "hip/hip_runtime.h"
// Vector dot product.
//
// The difference with dot-product.cu is that the kernel computes a single
// result instead of a block of partial results that must be reduced on the
// host.
#include <stdio.h>
#include <math.h>

#include "../common/cuda-error.h"

#include "lock.h"

#define imin(a, b) (a < b ? a : b)

#define sum_squares(x) (x*(x+1)*(2*x+1)/6)

// Number of float numbers (36 millions).
const int N = 33 * 1024 * 1024;

// Number of threads for each block.
const int THREADS = 256;

// By default there are 32 blocks, but if the source array is shorter there may
// be fewer than 32 blocks.
const int BLOCKS = imin(32, (N + THREADS - 1) / THREADS );

// c is volatile because it is read and written by thread of differents blocks,
// so the compiler must not optimize the access of this variable (eg. L1 cache).
__global__ void dot(mutex_t *m, float *a, float *b, volatile float *c) {
    __shared__ float cache[THREADS];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float tmp = 0;
    while (tid < N) {
        tmp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // Set cache value.
    cache[cacheIndex] = tmp;

    // Sync threads in the block.
    __syncthreads();

    // For reductions, threadsPerBlock must be a power of 2 because of the
    // following code
    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];

        __syncthreads();
        i /= 2;
    }

    // Only one thread of the block will write to c.
    if (cacheIndex == 0) {
        mutexLock(m);
        printf("Sum of block %d\n", blockIdx.x); // DEBUG.
        *c += cache[0];
        mutexUnlock(m);
    }
}

int main() {
    float *a, *b, c;
    float *dev_a, *dev_b, *dev_c;

    // Allocate memory host side.
    a = (float *) malloc(sizeof(float) * N);
    b = (float *) malloc(sizeof(float) * N);

    // Allocate memory device side.
    HANDLE_ERROR(hipMalloc((void **)&dev_a, sizeof(float) * N));
    HANDLE_ERROR(hipMalloc((void **)&dev_b, sizeof(float) * N));
    HANDLE_ERROR(hipMalloc((void **)&dev_c, sizeof(float)));
    HANDLE_ERROR(hipMemset(dev_c, 0, sizeof(float)));

    // Fill the input arrays.
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    // Create events and start timer.
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start));

    // Copy input array to the device.
    HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(float) * N,
                hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(float) * N,
                hipMemcpyHostToDevice));

    // Create mutex lock and launch kernel.
    mutex_t *mutex = mutexCreate();
    dot<<<BLOCKS, THREADS>>>(mutex, dev_a, dev_b, dev_c);

    // Copy output array to host.
    HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(float), hipMemcpyDeviceToHost));

    // Stop timer and get elapsed time.
    HANDLE_ERROR(hipEventRecord(stop));
    HANDLE_ERROR(hipEventSynchronize(stop));
    float elapsed;
    HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));

    printf("GPU Time: %3.1f ms\n", elapsed);
    printf("(GPU value) %.6g = %.6g (CPU value)\n", c,
            2 * sum_squares((float)(N-1)));

    // Free memory both host and device side.
    mutexDestroy(mutex);
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));
    free(a);
    free(b);

    return 0;
}
