#include "hip/hip_runtime.h"
#include <stdio.h>

#include "../common/cuda-error.h"

#define N 10

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x;

    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_a, sizeof(int)*N));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, sizeof(int)*N));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)*N));

    // Init arrays.
    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // Copy host arrays to device.
    HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(int)*N, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(int)*N, hipMemcpyHostToDevice));

    add<<<N, 1>>>(dev_a, dev_b, dev_c);

    // Copy device result array to host.
    HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int)*N, hipMemcpyDeviceToHost));

    // Display results.
    for (int i = 0; i < N; i++)
        printf("%2d + %2d = %2d\n", a[i], b[i], c[i]);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
