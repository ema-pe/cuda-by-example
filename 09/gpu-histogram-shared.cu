#include "hip/hip_runtime.h"
// Histogram calculation of a randomly generated data.
//
// This is the GPU version using global and shared memory atomics.
//
// There is one issue with this version: it generates random input data in the
// host and then copies it to the device.
#include <stdlib.h>
#include <stdio.h>

#include "cuda-error.h"

const size_t HISTO_SIZE = 256;

const size_t DATA_SIZE = 500 * 1024 * 1024; // 500 MB.

__global__ void histo_kernel(unsigned char *block, const size_t block_size, unsigned int *histogram) {
    // To reduce the concurrent access to the global histogram array, we use a
    // shared histogram (a copy shared for all threads of a single block). We
    // then merge the shared histograms into the global histogram.
    __shared__ unsigned int shared_hist[HISTO_SIZE];

    // This assumes each block has 256 linear threads and HISTO_SIZE == 256.
    shared_hist[threadIdx.x] = 0;

    // Wait all threads have initialized the shared histogram.
    __syncthreads();

    // Linearized offset in block.
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Total number of threads launched for this kernel.
    int tot_theads = blockDim.x * gridDim.x;

    while (i < block_size) {
        // Add atomically 1 to the histogram, it is needed because other threads
        // can access to the same address.
        atomicAdd(&(shared_hist[block[i]]), 1);

        // Move to the next block data.
        i += tot_theads;
    }

    // Wait all threads have updated the shared histogram.
    __syncthreads();

    // Each thread updates a global histogram entry. Since the kernel starts
    // with 256 threads, we cover all histogram entries.
    atomicAdd(&(histogram[threadIdx.x]), shared_hist[threadIdx.x]);
}

int main() {
    // Create events for timing.
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    // Input data.
    unsigned char *block = random_block(DATA_SIZE);

    // Allocate the data block on the device and copy the host version to it.
    unsigned char *dev_block;
    HANDLE_ERROR(hipMalloc((void **)&dev_block, DATA_SIZE));
    HANDLE_ERROR(hipMemcpy(dev_block, block, DATA_SIZE,
                hipMemcpyHostToDevice));

    // Allocate the histogram on the device and set it to zero.
    unsigned int *dev_histogram;
    HANDLE_ERROR(hipMalloc((void **)&dev_histogram, HISTO_SIZE * sizeof(unsigned int)));
    HANDLE_ERROR(hipMemset(dev_histogram, 0, HISTO_SIZE * sizeof(unsigned int)));

    // Call kernel.
    //
    // The block dimension is just a linear 256 threads, one for each histogram
    // bin.
    //
    // The grid dimension is the number of multiprocessor * 2.
    hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));

    // Equivalent.
    //dim3 grid(prop.multiProcessorCount*8);
    //dim3 block(HISTO_SIZE);
    int blocks = prop.multiProcessorCount * 8;
    int threads = HISTO_SIZE;
    printf("Blocks = %d\tThreads = %d\n", blocks, threads);
    histo_kernel<<<blocks, threads>>>(dev_block, DATA_SIZE, dev_histogram);

    // Copy the histogram from the device to the host.
    unsigned int histogram[256];
    HANDLE_ERROR(hipMemcpy(histogram, dev_histogram,
                HISTO_SIZE * sizeof(unsigned int), hipMemcpyDeviceToHost));

    // Stop timer.
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));

    // Calculate and print elapsed time.
    float elapsed;
    HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));
    printf("Time to generate: %3.1f ms\n", elapsed);

    // Check the calculate histogram is correct.
    for (int i = 0; i < DATA_SIZE; i++)
        histogram[block[i]]--;
    size_t total = 0;
    for (int i = 0; i < HISTO_SIZE; i++)
        total += histogram[i];

    if (total != 0) // Total should be zero if histogram is correct.
        printf("There are %ld elements that doesn't match the input block data\n", total);

    // Free all allocated data on both device and host.
    HANDLE_ERROR(hipFree(dev_block));
    HANDLE_ERROR(hipFree(dev_histogram));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
    free(block);

    return EXIT_SUCCESS;
}
