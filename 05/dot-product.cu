#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>

#include "../common/cuda-error.h"

#define imin(a, b) (a < b ? a : b)

#define sum_squares(x) (x*(x+1)*(2*x+1)/6)


const int N = 33 * 1024;

const int threadsPerBlock = 256;

// By default there are 32 blocks, but if the source array is shorter there may
// be fewer than 32 blocks.
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1) / threadsPerBlock);

__global__ void dot(float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float tmp = 0;
    while (tid < N) {
        tmp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // Set cache value.
    cache[cacheIndex] = tmp;

    // Sync threads in the block.
    __syncthreads();

    // For reductions, threadsPerBlock must be a power of 2 because of the
    // following code
    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];

        __syncthreads();
        i /= 2;
    }

    // Only one thread of the block will write to c.
    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

int main() {
    float *a, *b, c, *partial_c;
    float *dev_a, *dev_b, *dev_partial_c;

    // Allocate memory host side.
    a = (float *) malloc(sizeof(float) * N);
    b = (float *) malloc(sizeof(float) * N);
    partial_c = (float *) malloc(sizeof(float) * blocksPerGrid);

    // Allocate memory device side.
    HANDLE_ERROR(hipMalloc((void **)&dev_a, sizeof(float) * N));
    HANDLE_ERROR(hipMalloc((void **)&dev_b, sizeof(float) * N));
    HANDLE_ERROR(hipMalloc((void **)&dev_partial_c,
                sizeof(float) * blocksPerGrid));

    // Fill the input arrays.
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    // Copy input array to the device.
    HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(float) * N,
                hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(float) * N,
                hipMemcpyHostToDevice));

    dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_partial_c);

    // Copy output array to host.
    HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c,
                sizeof(float) * blocksPerGrid, hipMemcpyDeviceToHost));

    // Make last reduction on host.
    c = 0;
    for (int i = 0; i < blocksPerGrid; i++)
        c += partial_c[i];

    printf("(GPU value) %.6g = %.6g (CPU value)\n", c,
            2 * sum_squares((float)(N-1)));

    // Free memory both host and device side.
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_partial_c));
    free(a);
    free(b);
    free(partial_c);

    return 0;
}
