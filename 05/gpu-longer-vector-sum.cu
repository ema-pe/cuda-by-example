#include "hip/hip_runtime.h"
#include <stdio.h>

#include "../common/cuda-error.h"

#define N 1000

__global__ void add(int *a, int *b, int *c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    HANDLE_ERROR(hipMalloc((void**)&dev_a, sizeof(int)*N));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, sizeof(int)*N));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(int)*N));

    // Init arrays.
    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // Copy host arrays to device.
    HANDLE_ERROR(hipMemcpy(dev_a, a, sizeof(int)*N, hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, sizeof(int)*N, hipMemcpyHostToDevice));

    // 128 threads for each block.
    // Number of blocks depends on the array's size.
    add<<<(N+127)/128, 128>>>(dev_a, dev_b, dev_c);

    // Copy device result array to host.
    HANDLE_ERROR(hipMemcpy(c, dev_c, sizeof(int)*N, hipMemcpyDeviceToHost));

    // Display results.
    for (int i = 0; i < N; i++)
        printf("%2d + %2d = %2d\n", a[i], b[i], c[i]);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
