#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "cuda-error.h"

const size_t CHUNK_SIZE = 1024 * 1024; // 1 million of elements.

const size_t DATA_SIZE = CHUNK_SIZE * 20; // 20 million of elements.

// Just a dummy kernel to do some calculation based on array a and b and store
// the result on c.
__global__ void kernel( int *a, int *b, int *c ) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < CHUNK_SIZE) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;

        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;

        c[idx] = (as + bs) / 2;
    }
}

int main() {
    // Check if the device supports concurrent copy and kernel execution.
    hipDeviceProp_t prop;
    int device;
    HANDLE_ERROR(hipGetDevice(&device));
    HANDLE_ERROR(hipGetDeviceProperties(&prop, device));

    // prop.deviceOverlap is deprecated.
    printf("Device supports concurrent copy and kernel execution: ");
    switch (prop.asyncEngineCount) {
        case 0:
            puts("no");
            break;
        case 1:
            puts("yes (one direction)");
            break;
        case 2:
            puts("yes (both directions)");
            break;
    }

    // Initialize timers and start a timer.
    hipEvent_t start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start)); // Implicit stream = 0.

    // Create stream.
    hipStream_t stream;
    HANDLE_ERROR(hipStreamCreate(&stream));

    // Allocate memory on both host and device. On the host we allocate
    // page-locked memory because hipMemcpyAsync requires it.
    int *a, *b, *c, *dev_a, *dev_b, *dev_c;
    HANDLE_ERROR(hipMalloc((void **)&dev_a, CHUNK_SIZE * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&dev_b, CHUNK_SIZE * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **)&dev_c, CHUNK_SIZE * sizeof(int)));
    HANDLE_ERROR(hipHostAlloc((void **)&a, DATA_SIZE * sizeof(int),
                hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void **)&b, DATA_SIZE * sizeof(int),
                hipHostMallocDefault));
    HANDLE_ERROR(hipHostAlloc((void **)&c, DATA_SIZE * sizeof(int),
                hipHostMallocDefault));

    // Initialise the two input arrays.
    for (int i = 0; i < DATA_SIZE; i++) {
        a[i] = rand();
        b[i] = rand();
    }

    // The input arrays are too big: we split them into chunks and launch the
    // hipMemcpy and kernel for each chunk.
    //
    // Because hipMemcpyAsync and kernel startup are async, we have actually
    // appended the tasks to the single stream. After the for loop has finished,
    // we need to wait on the host side for the stream tasks to finish.
    for (int chunk = 0; chunk < DATA_SIZE; chunk += CHUNK_SIZE) {
        // Copy the chunks of input array to the device.
        HANDLE_ERROR(hipMemcpyAsync(dev_a, a + chunk, CHUNK_SIZE * sizeof(int),
                    hipMemcpyHostToDevice, stream));
        HANDLE_ERROR(hipMemcpyAsync(dev_a, a + chunk, CHUNK_SIZE * sizeof(int),
                    hipMemcpyHostToDevice, stream));

        // Launch the kernel.
        const int blocks = CHUNK_SIZE / 256;
        const int threads = 256;
        kernel<<<blocks, threads, 0, stream>>>(dev_a, dev_b, dev_c);

        // Copy the result chunk on the host.
        HANDLE_ERROR(hipMemcpyAsync(a + chunk, dev_a, CHUNK_SIZE * sizeof(int),
                    hipMemcpyDeviceToHost, stream));
    }
    HANDLE_ERROR(hipStreamSynchronize(stream));

    // Stop timer and get elapsed time.
    HANDLE_ERROR(hipEventRecord(stop)); // Implicit stream = 0.
    HANDLE_ERROR(hipEventSynchronize(stop));
    float elapsed;
    HANDLE_ERROR(hipEventElapsedTime(&elapsed, start, stop));
    printf("Time taken: %3.1f ms\n", elapsed);

    // Free all allocated memory.
    HANDLE_ERROR(hipStreamDestroy(stream));
    HANDLE_ERROR(hipFree(dev_a));
    HANDLE_ERROR(hipFree(dev_b));
    HANDLE_ERROR(hipFree(dev_c));
    HANDLE_ERROR(hipHostFree(a));
    HANDLE_ERROR(hipHostFree(b));
    HANDLE_ERROR(hipHostFree(c));
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    return EXIT_SUCCESS;
}
